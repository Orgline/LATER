#include "hip/hip_runtime.h"
#include "LATER.h"

hipEvent_t begin, end;
void startTimer()
{
    hipEventCreate(&begin);
    hipEventRecord(begin);
    hipEventCreate(&end);
}

float stopTimer()
{
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, begin, end);
    hipEventDestroy(begin);
    hipEventDestroy(end);
    return milliseconds;
}

__global__
void s2h(int m, int n, float *as, int ldas, __half *ah, int ldah)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		ah[i + j*ldah] = __float2half(as[i + j*ldas]);
	}
}

__global__
void h2s(int m, int n,__half *ah, int ldah, float *as, int ldas)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		as[i + j*ldah] = __half2float(ah[i + j*ldas]);
	}
}
