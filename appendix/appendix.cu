#include "LATER.h"

struct cudaCtxt {
	hipblasHandle_t cublas_handle;
	hipsolverHandle_t cusolver_handle;
};

hipEvent_t begin, end;
void startTimer()
{
    hipEventCreate(&begin);
    hipEventRecord(begin);
    hipEventCreate(&end);
}

float stopTimer()
{
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, begin, end);
    hipEventDestroy(begin);
    hipEventDestroy(end);
    return milliseconds;
}