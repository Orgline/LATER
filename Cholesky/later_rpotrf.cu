#include "hip/hip_runtime.h"
#include "LATER.h"

#include <hip/hip_fp16.h>

#include <assert.h>

#define BLOCKSIZE 2048
#define LWORK 65536

int chol_info;
int lwork;

int *dev_info;

float chol_panel = 0.0;
float chol_gemm = 0.0;

/*
This function performs recursive Cholesky factorization
*/

void l_potrf(cudaCtxt ctxt, int n, float* A, int lda, float* work, __half* hwork)
{
    float ms;
    //printf("n = %d\n", n);
    if(n<=BLOCKSIZE)
    {
        //printMatrixDeviceBlock("AAA.csv", n,n, A,lda);

        startTimer();
        hipsolverDnSpotrf(ctxt.cusolver_handle ,
            HIPBLAS_FILL_MODE_LOWER,
            n, A, lda,
            work, LWORK,
            dev_info);
        
        chol_panel += stopTimer();

        printf("panel takes %f ms\n", chol_panel);

        //assert(HIPSOLVER_STATUS_SUCCESS == status);
        //printf("status = %d\n", status);
        //printMatrixDeviceBlock("LLL.csv", n,n, A,lda);
        
        //hipMemcpy(&chol_info, dev_info, sizeof(int), hipMemcpyDeviceToHost);
        
        //printf("info = %d\n", chol_info);
        return;
    }

    l_potrf(ctxt, n/2, A, n, work, hwork);

    startTimer();

    later_rtrsm(ctxt.cublas_handle, 'l', 'r', 't', n/2, n/2, A, n, A+n/2, n, hwork);

    later_rsyrk(ctxt.cublas_handle, n/2, n/2, -1.0, A+n/2, n, 1.0, A+n/2*n+n/2, n, hwork);

    ms = stopTimer();
    
    chol_gemm+=ms;

    printf("n = %d,gemm takes %f ms, update flops is %f TFLOPS\n", n/2, chol_gemm, 2.0*n/2*n/2*n/2/ms/1e9);

    l_potrf(ctxt, n/2, A+n/2*n+n/2, n, work, hwork);
}

void later_rpotrf(char uplo, int n, float* A, int lda, float* work, __half* hwork)
{
    cudaCtxt ctxt;
    hipblasCreate(&ctxt.cublas_handle);
    hipsolverDnCreate(&ctxt.cusolver_handle);
    //printMatrixDeviceBlock("A.csv", n,n, A,n);
    hipMalloc(&dev_info, sizeof(int));

    if(uplo == 'l')
    {
        
        l_potrf(ctxt, n, A, lda, work, hwork);
    }

    printf("Panel takes %f ms, update takes %f ms\n", chol_panel, chol_gemm);

    printf("TFLOPS is %lf TFLOPS\n", 1.0/3.0*n*n*n/(chol_gemm+chol_panel)/1e9);

    hipblasDestroy(ctxt.cublas_handle);
    hipsolverDnDestroy(ctxt.cusolver_handle);

    //printMatrixDeviceBlock("L.csv", n, n, A, n);

    return; 
}