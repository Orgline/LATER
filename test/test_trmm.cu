#include "hip/hip_runtime.h"
#include "LATER.h"

#include <stdlib.h>

long m,n;
bool checkFlag;

int parseArguments(int argc,char *argv[])
{
	m = atoi(argv[1]);
	n = atoi(argv[2]);
	for (int i=3; i<argc; i++) {
		if(strcmp(argv[i], "-check") == 0) {
			checkFlag = true;
		}
	}
	return 0;
}

int main(int argc,char *argv[])
{
	if (argc < 3) {
		printf("Usage: test m n [options]\n");
		printf("Options:\n\t-check: enable checking the backward error\n");
		return 0;
	}
	if(parseArguments(argc,argv)!=0)
	{
		return 0;
	}
	float *A;
	hipMalloc(&A, sizeof(float)*m*m);
	float *B;
	hipMalloc(&B, sizeof(float)*m*n);

	__half *hwork;
	hipMalloc(&hwork, sizeof(__half)*(m/2*m/2+m/2*n));

	float *hA;
	hA = (float*)malloc(sizeof(float)*m*m);
	for(long i=0;i<m*m;i++)
	{
		hA[i] = 0.1;
	}
	hipMemcpy(A, hA, sizeof(float)*m*m, hipMemcpyHostToDevice);
	dim3 grid((m+31)/32, (m+31)/32);
	dim3 block(32,32);
	clearTri<<<grid, block>>>('u', m, m, A, m);

	float *hB;
	hB= (float*)malloc(sizeof(float)*m*n);

	for(long i=0;i<m*n;i++)
	{
		hB[i] = 1.0;
	}
	hipMemcpy(B, hB, sizeof(float)*m*n, hipMemcpyHostToDevice);

	float *C;
	hipMalloc(&C, sizeof(float)*m*n);
	float *tempC = A; //hipMalloc(&tempC, sizeof(float)* m *n);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	//printf("debug 1\n");
	float sone = 1.0;
	float snegone = -1.0;
	float szero = 0.0;

	float *work;
	hipMalloc(&work, sizeof(float)*m*n);
	//printf("debug 1\n");
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m,
			&sone, A, m, B, m,
			&szero, work, m
		   );

	hipMemcpy(B, work, sizeof(float)*m*n, hipMemcpyDeviceToDevice);

	float *dC;
	if(checkFlag)
	{
		//printf("Check forwards error\n");
		//float *dC;
		hipMalloc(&dC, sizeof(float)*m*n);

		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m,
				&sone, A, m, B, m,
				&szero, dC, m
			   );
	}


		later_rtrmm(m, n, A, m, B, m, C, m, tempC, hwork);


	if(checkFlag)
	{
		//printf("Check forwards error\n");
		//float *dC;
		/*	hipMalloc(&dC, sizeof(float)*m*n);

			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m,
			&sone, A, m, B, m,
			&szero, dC, m
			); 
		 */
		float normC = snorm(m, n, dC);
		printf("normC = %lf\n", normC);


		hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n,
				&sone,
				dC, m,
				&snegone,
				C, m,
				dC, m);


		printf("Forward error ||C-hat(C)||/||C|| is %.6e\n", snorm(m,n,dC)/normC);

		hipFree(dC);
	}


	{
		startTimer();
		hipblasStrmm(handle,
				HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
				HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
				m, n,
				&sone,
				A, m,
				B, m,
				C, m);
		printf("cuSOLVER strmm takes %lf\n", stopTimer());
	}
	hipFree(work);
	hipFree(hwork);
/*	float ms;
	startTimer();
	__half *hAA;
	hipMalloc(&hAA, sizeof(__half)*m*m);
	__half *hBB;*/
	//        hipMalloc(&hBB, sizeof(__half)*m*n);

	//startTimer();

/*	dim3 grid1((m+1)/32, (n+1)/32);
	dim3 block1(32,32);
	s2h<<<grid, block>>>(m, m, A, m, hAA, m); // I am not doing*/ 
	hipFree(A);

//	hipMalloc(&hBB, sizeof(__half)*m*n);
//	s2h<<<grid1, block1>>>(m, n, B, n,hBB, n); // I am not doing 
	hipFree(B);
/*	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, m,
			&sone, hAA, HIP_R_16F, m, hBB, HIP_R_16F, m,
			&szero, C, HIP_R_32F, m, HIP_R_32F,
			CUBLAS_GEMM_DEFAULT_TENSOR_OP
		    );*/
//	ms = stopTimer();

//	printf("NAVE TC-GEMM - TRMM takes %lf ms\n", ms);
	tempC = NULL;
	//	hipFree(A);
	//hipFree(B);
	hipFree(C);    
	//    hipFree(tempC);
	//	hipFree(work);
	//	hipFree(hwork);
	hipblasDestroy(handle);

	return 0;
}
