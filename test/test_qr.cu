#include "hip/hip_runtime.h"
#include "LATER.h"
#include "LATER_QR.h"
#include <assert.h>
#include <string>



#define NMIN 128

int algo;
int m,n;

//for rgsqrf
void checkResult(int m,int n,float* A,int lda, float *Q, int ldq, float *R, int ldr);
//for rhouqr
void checkResult(int m,int n, float *A, int lda, float *W, int ldw, float *Y, int ldy , float *R, int ldr);
void sgemm(int m,int n,int k,float *dA,int lda, float *dB,int ldb,float *dC, int ldc,float alpha,float beta);
void checkOtho(int,int,float*, int);

bool checkFlag = false;

int parseArguments(int argc,char *argv[])
{
    algo = atoi(argv[1]);
    m = atoi(argv[2]);
    n = atoi(argv[3]);
    for (int i=4; i<argc; i++) {
        if(strcmp(argv[i], "-check") == 0) {
            checkFlag = true;
        }
    }
    return 0;
}

int main(int argc,char *argv[])
{
    if (argc < 4) {
        printf("Usage: test algo m n [options]\n");
        printf("Options:\n\t-check: enable checking the orthogonality and backward error\n");
        return 0;
    }
    if(parseArguments(argc,argv)!=0)
    {
        return 0;
    }
    print_env();
    float *A;
    hipMalloc(&A,sizeof(float)*m*n);
    float *R;
    hipMalloc(&R,sizeof(float)*n*n);

    generateUniformMatrix(A,m,n);

    //printMatrixDeviceBlock("A.csv",m,n,A,m);


    float *dA;

    cudaCtxt ctxt {};
    hipblasCreate(&ctxt.cublas_handle );
    hipsolverDnCreate(&ctxt.cusolver_handle );

    int lwork;

    


        
    if (algo == 1)
    {

        //int lwork = (n/2)*(n/2);
        int lwork = m/256*32*n;
        float *work;
        __half *hwork;
        int lhwork = m*n;
        hipMalloc( &hwork, sizeof(__half) * lhwork );
        hipMalloc( &work, sizeof(float)*lwork );
        printf("Perform RGSQRF\nmatrix size %d*%d\n",m,n);
        //startTimer();

        later_rgsqrf(ctxt, m,n,A,m,R,n,work,lwork,hwork,lhwork);

        //float ms = stopTimer();
        //printf("RGSQRF takes %.0f ms, exec rate %.0f GFLOPS\n", ms, 
                //2.0*n*n*( m -1.0/3.0*n )/(ms*1e6));

        if (checkFlag) {


            checkOtho(m, n, A, m);

            hipMalloc(&dA,sizeof(float)*m*n);
            generateUniformMatrix(dA,m,n);

            checkResult(m, n, dA, m, A, m, R, n);
            hipFree(dA);
        }
        hipFree(work);
        hipFree(hwork);
    }

    if (algo == 2)
    {
        printf("Perform RHOUQR\nmatrix size %d*%d\n",m,n);
        __half *hwork;
        int lhwork = m*n;
        hipMalloc( &hwork, sizeof(__half) * lhwork );
        float *U;
        hipMalloc(&U,sizeof(float)*32*32);
        float *W;
        hipMalloc(&W,sizeof(float)*m*n);

        float *work;
        int lwork = m/256*32*n;
        hipMalloc(&work, sizeof(float)*lwork);

        //startTimer();
        later_rhouqr(m, n, A, m, W, m, R, n, work, lwork, hwork, lhwork, U);
        //float ms = stopTimer();
        //printf("RHOUQR takes %.0f ms, exec rate %.0f GFLOPS\n", ms, 
                //2.0*n*n*( m -1.0/3.0*n )/(ms*1e6));

        if(checkFlag)
        {
            hipMalloc(&dA,sizeof(float)*m*n);
            generateUniformMatrix(dA,m,n);
            later_ormqr(m, n, W, m, A, m, work);
            //printMatrixDeviceBlock("Q.csv",m,n,W,m);
            //checkResult( m, n, dA, m, W, m, A, m , R, n );
            checkOtho(m, n, W, m);
            dim3 grid96( (n+1)/32, (n+1)/32 );
            dim3 block96( 32, 32 );
            clearTri<<<grid96,block96>>>('l',n,n,R,n);  
            //printMatrixDeviceBlock("R.csv",n,n,R,n);
            checkResult(m, n, dA, m, W, m, R, n);
            hipFree(dA);
        }

        hipFree(U);
        hipFree(W);
        hipFree(work);
        hipFree(hwork);
    }

    if(algo == 3)
    {
        printf("Perform BHOUQR\nmatrix size %d*%d\n",m,n);
        __half *hwork;
        int lhwork = m*n;
        hipMalloc( &hwork, sizeof(__half) * lhwork );
        float *U;
        hipMalloc(&U,sizeof(float)*32*32);
        float *W;
        hipMalloc(&W,sizeof(float)*m*n);

        float *work;
        int lwork = m*n;
        hipMalloc(&work, sizeof(float)*lwork);

        //startTimer();
        later_bhouqr(m, n, A, m, W, m, R, n, work, lwork, hwork, lhwork, U);
        //float ms = stopTimer();
        //printf("BHOUQR takes %.0f ms, exec rate %.0f GFLOPS\n", ms, 
                //2.0*n*n*( m -1.0/3.0*n )/(ms*1e6));
        
        if(checkFlag)
        {
            hipMalloc(&dA,sizeof(float)*m*n);
            generateUniformMatrix(dA,m,n);
            later_ormqr2(m, n, W, m, A, m, work);
            checkOtho(m, n, W, m);
            checkResult(m, n, dA, m, W, m, R, n);
            hipFree(dA);
        }
        
        hipFree(U);
        hipFree(W);
        hipFree(work);
        hipFree(hwork);

    }

    //reference implementation in cuSOLVER
    {
        generateUniformMatrix(A,m,n);
        int lwork = 0;
        auto status = hipsolverDnSgeqrf_bufferSize(
                ctxt.cusolver_handle, m, n, A, m, &lwork);
        assert(HIPSOLVER_STATUS_SUCCESS == status);
        int *devInfo;
        hipMalloc((void**)&devInfo, sizeof(int));
        float *d_work, *d_tau;
        hipMalloc((void**)&d_work, sizeof(float)*lwork);
        hipMalloc((void**)&d_tau, sizeof(float)*m);
        startTimer();
        status = hipsolverDnSgeqrf( ctxt.cusolver_handle, m, n, A, m,
                d_tau, d_work, lwork, devInfo);
        assert(HIPSOLVER_STATUS_SUCCESS == status);
        float ms = stopTimer();
        printf("CUSOLVER SGEQRF takes %.0f ms, exec rate %.0f GFLOPS\n", ms,
               2.0*n*n*( m -1.0/3.0*n )/(ms*1e6));
        hipFree(d_work);
        hipFree(d_tau);
        hipFree(devInfo);
    }

    hipFree(A);
    hipFree(R);

//    hipFree(R);
//    hipFree(dA);
    return 0;
}

void checkResult(int m,int n,float* A,int lda, float *Q, int ldq, float *R, int ldr)
{
    float normA = snorm(m,n,A);
    float alpha = 1.0;
    float beta = -1.0;
    startTimer();
    sgemm(m,n,n,Q,ldq,R,ldr,A,lda,alpha,beta);
    float ms = stopTimer();
    printf("SGEMM m*n*k %d*%d*%d takes %.0f (ms), exec rate %.0f GFLOPS\n",
            m, n, n, ms, 2.0*m*n*n/(ms*1e6));
    float normRes = snorm(m,n,A);
    printf("Backward error: ||A-QR||/(||A||) = %.6e\n",normRes/normA);
}

void sgemm(int m,int n,int k,float *dA,int lda, float *dB,int ldb,float *dC, int ldc,float alpha,float beta)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float sone = alpha;
    float szero = beta;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
         m,n,k, 
         &sone, dA, lda, 
         dB, ldb, 
         &szero, dC, ldc
    );
    hipblasDestroy(handle);
}

void checkOtho(int m,int n,float *Q, int ldq)
{
    float *I;
    hipMalloc(&I,sizeof(float)*n*n);

    //printMatrixDeviceBlock("Q.csv",m,n,Q,m);
      
	dim3 grid96( (n+1)/32, (n+1)/32 );
	dim3 block96( 32, 32 );
    setEye<<<grid96,block96>>>( n, n, I, n);
    float snegone = -1.0;
    float sone  = 1.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m,
        &snegone, Q, HIP_R_32F, ldq, Q, HIP_R_32F, ldq,
        &sone, I, HIP_R_32F, n, HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT);
    
    float normRes = snorm(n,n,I);
    printf("||I-Q'*Q||/N = %.6e\n",normRes/n);
    hipFree(I);
    hipblasDestroy(handle);
}


void checkResult(int m,int n, float *A, int lda, float *W, int ldw, float *Y, int ldy , float *R, int ldr)
{
    float *I;
    hipMalloc(&I,sizeof(float)*n*n);
      
	dim3 grid96( (n+1)/32, (n+1)/32 );
	dim3 block96( 32, 32 );
    setEye<<<grid96,block96>>>( n, n, I, n);
    float snegone = -1.0;
    float sone  = 1.0;

    float *WI;
    hipMalloc(&WI, sizeof(float)*m*n);
    dim3 grid1( (m+1)/32, (n+1)/32 );
	dim3 block1( 32, 32 );
    setEye<<<grid1,block1>>>( m, n, WI, m);

    clearTri<<<grid96,block96>>>('l',n,n,R,ldr);   
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,m,n,n,
        &snegone,W,HIP_R_32F, ldw, Y, HIP_R_32F, ldy,
        &sone, WI, HIP_R_32F, m, HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT
    );

    
    
    float normWI= snorm(m,n,WI);
    printf("normWI = %f\n", normWI);
    

    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m,
        &snegone, WI, HIP_R_32F, m, WI, HIP_R_32F, m,
        &sone, I, HIP_R_32F, n, HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT);
    
    float normRes = snorm(n,n,I);
   
    printf("||I-Q'*Q||/N = %.6e\n",normRes/n);

    //printMatrixDeviceBlock("AA.csv",m,n,A,lda);
    float normA = snorm(m,n,A);
    printf("normA = %f\n", normA);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, n,
        &snegone, WI, HIP_R_32F, m, R, HIP_R_32F, ldr,
        &sone, A, HIP_R_32F, lda, HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT
        );
    normRes = snorm(m,n,A);
    printf("||A-QR||/||A|| = %.6e\n",normRes/normA);
    hipFree(I);
    hipFree(WI);
}
