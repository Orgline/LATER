#include "hip/hip_runtime.h"
#include "LATER.h"

#include <stdlib.h>

long n, k;
bool checkFlag;

int parseArguments(int argc,char *argv[])
{
    n = atoi(argv[1]);
    k = atoi(argv[2]);
    for (int i=3; i<argc; i++) {
        if(strcmp(argv[i], "-check") == 0) {
            checkFlag = true;
        }
    }
    return 0;
}

int main(int argc,char *argv[])
{
    if (argc < 3) {
        printf("Usage: test m n [options]\n");
        printf("Options:\n\t-check: enable checking the backward error\n");
        return 0;
    }
    if(parseArguments(argc,argv)!=0)
    {
        return 0;
    }
    float *A;
    hipMalloc(&A, sizeof(float)*n*k);

    float *C;
    hipMalloc(&C, sizeof(float)*n*n);

    __half *hwork;
    hipMalloc(&hwork, sizeof(__half)*n*k);

    generateUniformMatrix(A,n,k);

    //startTimer();
    later_rsyrk(n, k, A, n, C, n, hwork);
    float ms;

    float *tC;
    hipMalloc(&tC, sizeof(float)*n*n);

    hipMemcpy(tC, C, sizeof(float)*n*n, hipMemcpyDeviceToDevice);



    
    //printf("SYRK takes %lfms\n", ms);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float sone  = 1.0;
    float szero = 0.0;
    startTimer();
    
    hipblasSsyrk(handle,
        HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
        n, k,
        &sone,
        A, n,
        &szero,
        C, n
    );
    
    ms = stopTimer();

    printf("SYRK takes %lfms\n", ms);

    if(checkFlag)
    {
        float snegone = -1.0;
        hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n,
            &sone, C, n, &snegone, tC, n,
            C, n
        );
        printf("Forward error is %.6e\n",snorm(n,n,C)/snorm(n , n, tC));
    }

    startTimer();

    __half *Ah = hwork;

    dim3 grid((n+1)/32, (k+1)/32);
    dim3 block(32,32);
    s2h<<<grid, block>>>(n/2, k, A, n, Ah, n);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, k,
        &sone, Ah, HIP_R_16F, n, Ah, HIP_R_16F, n,
        &szero, C, HIP_R_32F, n, HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );

    ms = stopTimer();

    printf("TC-GEMM takes %lf ms, %lf TFLOPS\n", ms, 2.0*n*n*k/1e9/ms);

    

    hipblasDestroy(handle);

    hipFree(A);
    hipFree(hwork);
    hipFree(C);
    hipFree(tC);
}

