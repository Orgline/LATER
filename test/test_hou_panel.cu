#include "LATER.h"
#include "LATER_QR.h"

int main(int argc, char* argv[])
{
    float *A;
    int m=1024;
    int n=32;
    hipMalloc(&A, sizeof(float)*m*n);
    generateUniformMatrix(A, m, n);
    int nb = (m+255)/256;
    int r = m%256;
    int ldwork = m/256*32+32;
    int mm = m/256*32+32;
    int lda=m;

    printMatrixDeviceBlock<float>("A.csv",m,n,A,lda);

    float *R;
    int ldr = n*nb;
    hipMalloc(&R, sizeof(float)*n*n*nb);

    {
        startTimer();
        auto blockdim = dim3(32, 32);
        int nb = (m+255)/256;

        hou_kernel<256,32><<<nb, blockdim>>>(m, n, A, lda, R, ldr);
        float ms = stopTimer();
        CHECK_KERNEL();
        printf("%dx%d hou_kernel block takes %.3f (ms)\n", m, n, ms);
    }
    printMatrixDeviceBlock("Q.csv", m, n, A, lda);
    printMatrixDeviceBlock("R.csv", n*nb, n, R, ldr);
    generateUniformMatrix(A, m, n);

    {
        startTimer();
        auto blockdim = dim3(32, 32);
        mgs_kernel<<<1, 256>>>(m, n, A, lda, R, n);
        //mgs_kernel2<<<1, blockdim>>>(m, n, A,  lda, R, n);
        float ms = stopTimer();
        printf("%dx%d mgs_kernel block takes %.3f (ms)\n", m, n, ms);
    }



}