#include "hip/hip_runtime.h"
#include "LATER.h"
#include <hip/hip_fp16.h>
// #include "OC_gemm.h"
#include <random>
#include <assert.h>
#include <stdio.h>

__global__
void generateSyMatrix(int m, int n, float* dA,int lda, float *tmpA){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if (i<m && j<n) {
        tmpA[i+j*lda] = (dA[i+j*lda] + dA[j+i*lda])/2.0f;
    }

    __syncthreads();

    if (i<m && j<n) {
        dA[i+j*lda] = tmpA[i+j*lda];
    }
}


int main(int argc,char *argv[]){
	if (argc<2) exit(1);

	int n = atoi(argv[1]);
	int nb = atoi(argv[2]);

	cudaCtxt ctxt;
	hipblasCreate(&ctxt.cublas_handle );
	hipsolverDnCreate(&ctxt.cusolver_handle );

	print_env();

	float *A;
	hipMalloc(&A,sizeof(float)*n*n);
	int lda=n;    
	float *H;
	hipMalloc(&H, sizeof(float)*n*n);
	generateUniformMatrix(A, n ,n);
	dim3 grid1((n+31)/32,(n+31)/32);
	dim3 block1(32,32);
	generateSyMatrix<<<grid1, block1>>>(n,n,A,lda,H);
	printMatrixDeviceBlock("A_orig_block.csv", n, n, A, lda);

	float* AA;
	hipMalloc(&AA,sizeof(float)*n*n);
	hipMemcpy(AA, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice); 

	int lhwork=n*nb;
	int lwork=n*nb;

   	float *work;
    hipMalloc(&work,sizeof(float)*n*nb+nb*nb);
	hipMemset(work, 0, sizeof(float)*n*nb+nb*nb);
    __half *hwork;
    hipMalloc(&hwork,sizeof(__half)*n*nb+nb*nb);

	float* Dummy;
	hipMalloc(&Dummy,sizeof(float)*n*n);
	hipMemcpy(Dummy, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice); 
	later_rhouqr(n, nb, Dummy, n, work, n, work, nb, work, lwork, hwork, lhwork, work);
	
	// startTimer();
	ssytrd_sy2sb(ctxt, n, nb, A, AA, lda, work, lwork, hwork, lhwork);
	// float ms=stopTimer();
	// printf("SY2SB takes %f ms\n", ms);

	hipFree(A);
	hipFree(H);
	hipFree(hwork);
	hipFree(work);
	hipblasDestroy(ctxt.cublas_handle);
	hipsolverDnDestroy(ctxt.cusolver_handle);
	return 0;
}
