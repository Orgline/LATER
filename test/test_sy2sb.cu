#include "hip/hip_runtime.h"
#include "LATER.h"
#include <hip/hip_fp16.h>
// #include "OC_gemm.h"
#include <random>
#include <assert.h>
#include <stdio.h>

__global__
void generateSyMatrix(int m, int n, float* dA,int lda, float *tmpA){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i<m && j<n) {
        tmpA[i+j*lda] = (dA[i+j*lda] + dA[j+i*lda])/2.0f;
    }
    __syncthreads();
    if (i<m && j<n) {
        dA[i+j*lda] = tmpA[i+j*lda];
    }
}


int main(int argc,char *argv[]){
	if (argc<2) exit(1);

	int n = atoi(argv[1]);
	int nb = atoi(argv[2]);

	print_env();
	cudaCtxt ctxt;
	hipblasCreate(&ctxt.cublas_handle );
	hipsolverDnCreate(&ctxt.cusolver_handle );

	float *A;
	hipMalloc(&A,sizeof(float)*n*n);  
	float* AA;
	hipMalloc(&AA,sizeof(float)*n*n);  
	float *work;
    hipMalloc(&work,sizeof(float)*n*n*2);
    __half *hwork;
    hipMalloc(&hwork,sizeof(__half)*n*n*2);

	int lda=n;
	int lhwork=n*nb;
	int lwork=n*nb;

	generateUniformMatrix(A, n ,n);
	dim3 grid1((n+31)/32,(n+31)/32);
	dim3 block1(32,32);
	generateSyMatrix<<<grid1, block1>>>(n,n,A,lda,work);
	hipMemcpy(AA, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice);
	printMatrixDeviceBlock("A_orig_block.csv", n, n, A, lda); 


	float* Dummy;
	hipMalloc(&Dummy,sizeof(float)*n*n);
	dim3 grid2((n*2+31)/32,(n+31)/32);
	setInitialValue<<<grid2, block1>>>(2*n, n, work, 2*n, 0);
	hipMemcpy(Dummy, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice); 
	later_rhouqr(n, nb, Dummy, n, work, n, work, nb, work, lwork, hwork, lhwork, work);
	
	// startTimer();
	setInitialValue<<<grid1, block1>>>(2*n, n, work, 2*n, 0);
	ssytrd_sy2sb(ctxt, n, nb, A, AA, lda, work, lwork, hwork, lhwork);
	// float ms=stopTimer();
	// printf("SY2SB takes %f ms\n", ms);

	hipFree(A);
	// hipFree(H);
	hipFree(hwork);
	hipFree(work);
	hipblasDestroy(ctxt.cublas_handle);
	hipsolverDnDestroy(ctxt.cusolver_handle);
	return 0;
}
