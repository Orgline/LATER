#include "hip/hip_runtime.h"
#include "LATER.h"
#include <hip/hip_fp16.h>
// #include "OC_gemm.h"
#include <random>
#include <assert.h>
#include <stdio.h>

__global__
void generateSyMatrix(int m, int n, float* dA,int lda, float *tmpA){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if (i<m && j<n) {
        tmpA[i+j*lda] = (dA[i+j*lda] + dA[j+i*lda])/2.0f;
    }

    __syncthreads();

    if (i<m && j<n) {
        dA[i+j*lda] = tmpA[i+j*lda];
    }
}


int main(int argc,char *argv[]){
	if (argc<2) exit(1);

	int n = atoi(argv[1]);
	int nb = atoi(argv[2]);

	cudaCtxt ctxt;
	hipblasCreate(&ctxt.cublas_handle );
	hipsolverDnCreate(&ctxt.cusolver_handle );

	print_env();

	float *A;
	hipMalloc(&A,sizeof(float)*n*n);
	int lda=n;    
	float *H;
	hipMalloc(&H, sizeof(float)*n*n);
	generateUniformMatrix(A, n ,n);
	dim3 grid1((n+31)/32,(n+31)/32);
	dim3 block1(32,32);
	generateSyMatrix<<<grid1, block1>>>(n,n,A,lda,H);
	printMatrixDeviceBlock("A_orig.csv", n, n, A, lda);

	float* AA;
	hipMalloc(&AA,sizeof(float)*n*n);
	hipMemcpy(AA, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice); 

	// float *W;
	// dim3 grid2((n+31)/32,(nb+31)/32);
	// hipMalloc(&W,sizeof(float)*n*nb);
	// setInitialValue<<<grid2, block1>>>( n,nb,W,n,0.0);

	// float *Z;
	// hipMalloc(&Z,sizeof(float)*n*nb);
	// setInitialValue<<<grid2, block1>>>( n, nb, Z, n, 0.0);

	__half *hwork;
	int lhwork = n*nb;
	hipMalloc( &hwork, sizeof(__half)*n*n*2);

	// float *U;
	// hipMalloc(&U,sizeof(float)*nb*nb);
	// dim3 grid3((nb+31)/32,(nb+31)/32);
	// setInitialValue<<<grid3, block1>>>( nb, nb, U, nb, 0.0);

	
	// float *R;
	// hipMalloc(&R,sizeof(float)*nb*nb);
	// setInitialValue<<<grid3, block1>>>( nb, nb, R, nb, 0.0);
	
	float *work;
	int lwork = n*nb;
	hipMalloc(&work, sizeof(float)*n*n*2);
	dim3 grid2((n*2+31)/32,(n+31)/32);
	setInitialValue<<<grid2, block1>>>( 2*n, n, work, 2*n, 0.0);

	float* Dummy;
	hipMalloc(&Dummy,sizeof(float)*n*n);
	hipMemcpy(Dummy, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice); 
	later_rhouqr(n, nb, Dummy, n, work, n, work, nb, work, lwork, hwork, lhwork, work);
	
	// startTimer();
	ssytrd_sy2sb(ctxt, n, nb, A, AA, lda, work, lwork, hwork, lhwork);
//	float ms=stopTimer();
//	printf("SY2SB takes %f ms\n", ms);

	hipFree(A);
	hipFree(H);
	hipFree(hwork);
	hipFree(work);
	// hipFree(U);
	// hipFree(W);
	// hipFree(R);
	hipblasDestroy(ctxt.cublas_handle);
	hipsolverDnDestroy(ctxt.cusolver_handle);
	return 0;
}
