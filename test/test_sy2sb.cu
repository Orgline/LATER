#include "hip/hip_runtime.h"
#include "LATER.h"
//#include "LATER_QR.h"
#include <hip/hip_fp16.h>
#include "OC_gemm.h"
#include <random>
#include <assert.h>
#include <stdio.h>

__global__
void generateSyMatrix(int m, int n, float* dA,int lda, float *tmpA){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if (i<m && j<n) {
        tmpA[i+j*lda] = (dA[i+j*lda] + dA[j+i*lda])/2.0f;
    }

    __syncthreads();

    if (i<m && j<n) {
        dA[i+j*lda] = tmpA[i+j*lda];
    }
}


int main(int argc,char *argv[]){
    	if (argc<2) exit(1);

	int n = atoi(argv[1]);
    	int nb = atoi(argv[2]);

	cudaCtxt ctxt;
    	hipblasCreate(&ctxt.cublas_handle );
    	hipsolverDnCreate(&ctxt.cusolver_handle );

	float *A;
    	hipMalloc(&A,sizeof(float)*n*n);
	int lda=n;    
	float *H;
	hipMalloc(&H, sizeof(float)*n*n);
	generateUniformMatrix(A, n ,n);

	dim3 grid1((n+31)/32,(n+31)/32);
	dim3 block1(32,32);
	generateSyMatrix<<<grid1, block1>>>(n,n,A,lda,H);
//	printMatrixDeviceBlock("A.csv", n, n, A, lda);

	float* AA;
        hipMalloc(&AA,sizeof(float)*n*n);
        hipMemcpy(AA, A, sizeof(float)*n*n, hipMemcpyDeviceToDevice); 
	__half *hwork;
        int lhwork = n*n;
        hipMalloc( &hwork, sizeof(__half) * lhwork );
        float *U;
        hipMalloc(&U,sizeof(float)*nb*nb);
        float *W;
        hipMalloc(&W,sizeof(float)*n*n);
        float *R;
        hipMalloc(&R,sizeof(float)*nb*nb);
	float *Z;
        hipMalloc(&Z,sizeof(float)*n*n);
	float *work;
        int lwork = n/256*32*n;
        hipMalloc(&work, sizeof(float)*lwork);
//	startTimer();
	ssytrd_sy2sb(ctxt, n, nb, A, AA, lda, U, nb, W, n, R, nb, Z, n, work, lwork, hwork, lhwork);
//	float ms=stopTimer();
//	printf("SY2SB takes %f ms\n", ms);

	hipFree(A);
	hipFree(H);
	hipFree(hwork);
	hipFree(U);
	hipFree(W);
	hipFree(R);
	hipblasDestroy(ctxt.cublas_handle);
    	hipsolverDnDestroy(ctxt.cusolver_handle);
	return 0;
}
