#include "LATER.h"
#include "LATER_QR.h"

int main(int argc, char* argv[])
{
    float *A;
    int m=4096; // 32*256
    int n=32;
    hipMalloc(&A, sizeof(float)*m*n);
    generateUniformMatrix(A, m, n);
    int nb = (m+255)/256;
    int r = m%256;
    int ldwork = m/256*32+32;
    int mm = m/256*32+32;
    int lda=m;

    print_env();

    printMatrixDeviceBlock<float>("A.csv",m,n,A,lda);

    float *R;
    int ldr = n;
    hipMalloc(&R, sizeof(float)*n*n*nb);
    cudaCtxt ctxt;
    hipblasCreate(&ctxt.cublas_handle );
    hipsolverDnCreate(&ctxt.cusolver_handle );
    {
        float *work;
        hipMalloc(&work, 2*sizeof(float)*m*n);
        startTimer();
        hou_caqr_panel<256,32>(ctxt, m, n, A, lda, R, ldr, work);
        float ms = stopTimer();
        CHECK_KERNEL();
        printf("%dx%d hou_caqr_panel_256x32 block takes %.3f (ms)\n", m, n, ms);
        hipFree(work);
    }
    printMatrixDeviceBlock("Q.csv", m, n, A, lda);
    printMatrixDeviceBlock("R.csv", n, n, R, ldr);

    {
        float *work;
        hipMalloc(&work, 2*sizeof(float)*m*n);
        cudaCtxt ctxt;
        hipblasCreate(&ctxt.cublas_handle );
        hipsolverDnCreate(&ctxt.cusolver_handle );


        startTimer();
        mgs_caqr_panel_256x32(ctxt, m, n, A, lda, R, ldr, work);
//        ( m, n, A, lda, R, ldr);
        float ms = stopTimer();
        CHECK_KERNEL();
        printf("%dx%d mgs_caqr_panel_256x32 block takes %.3f (ms)\n", m, n, ms);
        hipFree(work);
    }

}