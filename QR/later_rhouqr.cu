#include "hip/hip_runtime.h"
#include "LATER.h"
#include "LATER_QR.h"

#include <hip/hip_fp16.h>

#define NMIN 32

/*
This routine performs recursive Householder QR factorization

The input A stores the original matrix A to be factorized
The output A stores the Householder vectors Y
The output W stores the W matrix of WY representation
The orthogonal matrix Q 
THe output R stor
*/

void qr(cudaCtxt ctxt, int m, int n, float *A, int lda, float *W, int ldw, float *R, int ldr, float *work, int lwork,
    __half *hwork, int lhwork, float* U);

void later_rhouqr(int m, int n, float* A, int lda, float* W, int ldw, float* R, int ldr, float* work, int lwork, __half* hwork, int lhwork, float* U)
{
    printf("Function rhouqr\n");
    
    cudaCtxt ctxt;
    hipblasCreate( & ctxt.cublas_handle );
    hipsolverDnCreate( & ctxt.cusolver_handle );

    qr(ctxt, m, n, A, lda, W, ldw, R, ldr, work, lwork, hwork, lhwork, U);
    
    hipblasDestroy(ctxt.cublas_handle);
    hipsolverDnDestroy(ctxt.cusolver_handle);
    return;
}

void qr(cudaCtxt ctxt, int m, int n, float *A, int lda, float *W, int ldw, float *R, int ldr, float *work, int lwork, __half *hwork, int lhwork, float* U)
{
    if(n<=NMIN)
    {
        return;
    }
}