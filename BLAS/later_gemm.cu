#include "hip/hip_runtime.h"
#include "OC_gemm.h"
#define cudaChk(stat)                                                                              \
    { cudaErrCheck_((stat), __FILE__, __LINE__); }
static void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
        exit(1);
    }
}
const static char *cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
        return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}
#define cublasChk(stat)                                                                            \
    { cublasErrCheck_((stat), __FILE__, __LINE__); }
static void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %s %s %d\n", cublasGetErrorString(stat), file, line);
        exit(1);
    }
}

static int is_mul_overflow(int a, int b) {
    if (a >= 0 && b >= 0) {
        return INT_MAX / a < b;
    } else if (a < 0 && b < 0) {
        return INT_MAX / a > b;
    } else if (a * b == INT_MIN) {
        return 0;
    } else {
        return a < 0 ? is_mul_overflow(-a, b) : is_mul_overflow(a, -b);
    }
}

size_t OC_gemm::tile_size() {
    auto free_mem_stream = mem_limit / num_stream;
    int i = 1;
    size_t mem_per_stream = 0;
    do {
        tm = M / i;
        tn = N / i;
        i *= 2;
        if (is_mul_overflow(tm, K) || is_mul_overflow(tn, K)) continue;
        mem_per_stream = (size_t(tm * K) + size_t(tm * tn) + size_t(K * tn)) * sizeof(float) +
                         (size_t(tm * K) + size_t(K * tn)) * sizeof(half);
        if (mem_per_stream < free_mem_stream) break;
    } while (true);
    return mem_per_stream;
}

OC_gemm::OC_gemm(int _M, int _N, int _K, std::shared_ptr<Mem_pool> _pool, size_t _mem_limit,
                 int _num_stream)
    : M(_M), N(_N), K(_K), pool(_pool), mem_limit(_mem_limit), num_stream(_num_stream),
      streams(_num_stream), handles(_num_stream), A_tiles(_num_stream), B_tiles(_num_stream),
      C_tiles(_num_stream), fA_tiles(_num_stream), fB_tiles(_num_stream) {
    mem_limit = mem_limit == 0 ? (pool->capacity() - pool->size()) : mem_limit;
    for (int i = 0; i < num_stream; i++) {
        cudaChk(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
        cublasChk(hipblasCreate(&handles[i]));
        cublasChk(hipblasSetStream(handles[i], streams[i]));
        cublasChk(hipblasSetMathMode(handles[i], HIPBLAS_TENSOR_OP_MATH));
    }
    auto mem_per_stream = tile_size();
    // std::cout << tm << "\t" << tn << "\n";
    for (int i = 0; i < num_stream; i++) {
        fA_tiles[i] = reinterpret_cast<float *>(pool->allocate(sizeof(float) * tm * K));
        fB_tiles[i] = reinterpret_cast<float *>(pool->allocate(sizeof(float) * tn * K));
        C_tiles[i] = reinterpret_cast<float *>(pool->allocate(sizeof(float) * tn * tm));
        A_tiles[i] = reinterpret_cast<half *>(pool->allocate(sizeof(half) * tm * K));
        B_tiles[i] = reinterpret_cast<half *>(pool->allocate(sizeof(half) * tm * K));
    }
}

OC_gemm::~OC_gemm() {
    for (int i = 0; i < num_stream; i++) {
        cudaChk(hipStreamDestroy(streams[i]));
        cublasChk(hipblasDestroy(handles[i]));
        pool->free(fA_tiles[i]);
        pool->free(fB_tiles[i]);
        pool->free(C_tiles[i]);
        pool->free(A_tiles[i]);
        pool->free(B_tiles[i]);
    }
}

void OC_gemm::gemm(hipblasOperation_t transa, hipblasOperation_t transb, const float &alpha,
                   const half *A, int lda, const half *B, int ldb, const float &beta, float *C,
                   int ldc) {
    int tlda = transa == HIPBLAS_OP_N ? tm : K;
    int tldb = transb == HIPBLAS_OP_N ? K : tn;
    for (size_t i = 0; i < (M / tm); i++) {
        const auto stream_id = i % num_stream;
        auto stream = streams[stream_id];
        if (transa == HIPBLAS_OP_N) {
            cublasChk(hipblasSetMatrixAsync(tm, K, sizeof(half), &A[i * tm], M, A_tiles[stream_id],
                                           tlda, stream));

        } else {
            cublasChk(hipblasSetMatrixAsync(K, tm, sizeof(half), &A[i * tm * K], K,
                                           A_tiles[stream_id], tlda, stream));
        }
        for (size_t j = 0; j < (N / tn); j++) {
            if (transb == HIPBLAS_OP_N) {
                cublasChk(hipblasSetMatrixAsync(K, tn, sizeof(half), &B[j * tn * K], K,
                                               B_tiles[stream_id], tldb, stream));
            } else {
                cublasChk(hipblasSetMatrixAsync(tn, K, sizeof(half), &B[j * tn], N,
                                               B_tiles[stream_id], tldb, stream));
            }
            cublasChk(hipblasGemmEx(handles[stream_id], transa, transb, tm, tn, K, &alpha,
                                   A_tiles[stream_id], HIP_R_16F, tlda, B_tiles[stream_id],
                                   HIP_R_16F, tldb, &beta, C_tiles[stream_id], HIP_R_32F, tm,
                                   HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            cublasChk(hipblasGetMatrixAsync(tm, tn, sizeof(float), C_tiles[stream_id], tm,
                                           &C[j * tn * M + i * tm], M, stream));
            cudaChk(hipMemsetAsync(C_tiles[stream_id], 0, tm * tn * sizeof(float), stream));
        }
    }
    cudaChk(hipDeviceSynchronize());
}

__global__ void __kernel_AB2half(const float *__restrict__ fA, const float *__restrict__ fB,
                                 half *__restrict__ hA, half *__restrict__ hB, const int size_A,
                                 const int size_B) {
    const auto tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (auto i = tid; i < size_A; i += gridDim.x * blockDim.x)
        hA[i] = __float2half(fA[i]);
    for (auto i = tid; i < size_B; i += gridDim.x * blockDim.x)
        hB[i] = __float2half(fB[i]);
}

void OC_gemm::gemm(hipblasOperation_t transa, hipblasOperation_t transb, const float &alpha,
                   const float *A, int lda, const float *B, int ldb, const float &beta, float *C,
                   int ldc) {

    int tlda = transa == HIPBLAS_OP_N ? tm : K;
    int tldb = transb == HIPBLAS_OP_N ? K : tn;
    for (size_t i = 0; i < (M / tm); i++) {
        const auto stream_id = i % num_stream;
        auto stream = streams[stream_id];
        if (transa == HIPBLAS_OP_N) {
            cublasChk(hipblasSetMatrixAsync(tm, K, sizeof(float), &A[i * tm], M, A_tiles[stream_id],
                                           tlda, stream));

        } else {
            cublasChk(hipblasSetMatrixAsync(K, tm, sizeof(float), &A[i * tm * K], K,
                                           A_tiles[stream_id], tlda, stream));
        }
        for (size_t j = 0; j < (N / tn); j++) {
            if (transb == HIPBLAS_OP_N) {
                cublasChk(hipblasSetMatrixAsync(K, tn, sizeof(float), &B[j * tn * K], K,
                                               B_tiles[stream_id], tldb, stream));
            } else {
                cublasChk(hipblasSetMatrixAsync(tn, K, sizeof(float), &B[j * tn], N,
                                               B_tiles[stream_id], tldb, stream));
            }
            auto size_A = tm * K;
            auto size_B = tn * K;
            __kernel_AB2half<<<(std::max(size_A, size_B) / 1024) + 1, 1024, 0, stream>>>(
                fA_tiles[stream_id], fB_tiles[stream_id], A_tiles[stream_id], B_tiles[stream_id],
                size_A, size_B);
            cublasChk(hipblasGemmEx(handles[stream_id], transa, transb, tm, tn, K, &alpha,
                                   A_tiles[stream_id], HIP_R_16F, tlda, B_tiles[stream_id],
                                   HIP_R_16F, tldb, &beta, C_tiles[stream_id], HIP_R_32F, tm,
                                   HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            cublasChk(hipblasGetMatrixAsync(tm, tn, sizeof(float), C_tiles[stream_id], tm,
                                           &C[j * tn * M + i * tm], M, stream));
            cudaChk(hipMemsetAsync(C_tiles[stream_id], 0, tm * tn * sizeof(float), stream));
        }
    }
    cudaChk(hipDeviceSynchronize());
}
/*

#ifdef TEST_OC

template <typename T> void prt(T *arr, int size) {
    for (int i = 0; i < size; i++)
        std::cout << arr[i] << ", ";
    puts("");
}
int main(int ac, char **av) {
    if (ac < 4) puts("Usage: ./a.out m n k [allowed GPU mem (MiB)]");
    int m = atoi(av[1]);
    int n = atoi(av[2]);
    int k = atoi(av[3]);

    size_t mem, free, total;
    cudaChk(hipMemGetInfo(&free, &total));
    mem = free;
    if (ac > 4) {
        mem = atol(av[4]) * 1024 * 1024;
    }
    std::cout << mem << "\t" << free << "\t" << total << std::endl;
    assert(mem <= free);

    const size_t elements = size_t(m) * size_t(k) + size_t(n) * size_t(k);
    std::vector<half> h_data(elements);
    std::vector<float> f_data(elements);
    std::uniform_real_distribution<float> distribution(0.0f, 2.0f);
    std::mt19937 engine;
    const auto f_data_size = f_data.size();
#pragma omp parallel for
    for (size_t i = 0; i < f_data_size; i++) {
        f_data[i] = distribution(engine);
        h_data[i] = __float2half(f_data[i]);
    }
    auto hA = h_data.data();
    auto hB = &h_data.data()[size_t(m) * size_t(k)];
    auto fA = f_data.data();
    auto fB = &f_data.data()[size_t(m) * size_t(k)];
    std::vector<float> C_data(size_t(m) * size_t(n), 0.0f);
    auto C = C_data.data();

    float alpha = 1.0f;
    float beta = 1.0f;

    auto pool = std::make_shared<Mem_pool>(mem); // Create memory pool
    /*
     * Usage:
     *  allocate memory: float* p = reinterpret_cast<float *>(pool->allocate(size));
     *  free memory: pool->free(p);
     */

/*
    OC_gemm OC(m, n, k, pool);
    puts("Created");
    OC.gemm(HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, hA, m, hB, k, beta, C, m);
    std::cout << C[0] << std::endl;
    OC.gemm(HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, fA, m, fB, k, beta, C, m);
    std::cout << C[0] << std::endl;
}
#endif*/