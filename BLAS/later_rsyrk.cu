#include "hip/hip_runtime.h"
#include "LATER.h"

#include <hip/hip_fp16.h>

#define BLOCKSIZE 2048

float pan = 0.0;
float ge = 0.0;



void syrk(hipblasHandle_t handle, int n, int k, float alpha, float *A, int lda, float beta, float *C, int ldc, __half *hwork)
{
    //printf("n = %d\n", n);
    //gpuErrchk( hipPeekAtLastError() );
    float sone  = 1.0;
    float szero = 0.0;
    if(n<=BLOCKSIZE)
    {
        //printf("alpha = %f, beta = %f\n", alpha, beta);
        //printMatrixDeviceBlock("A.csv", n ,k ,A, lda);
        //printMatrixDeviceBlock("C.csv", n ,n ,C, ldc);
        //startTimer();
        hipblasSsyrk(handle,
            HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
            n, k,
            &alpha,
            A, lda,
            &beta,
            C, ldc
        );
        //printMatrixDeviceBlock("fC.csv", n ,n ,C, ldc);

        //pan+=stopTimer();
        return;
    }
    syrk(handle, n/2, k, alpha, A, lda, beta, C, ldc, hwork);
    
    //startTimer();
    __half *Ah = hwork;
    __half *Bh = hwork+n/2*k;

    dim3 grid((n/2+1)/32, (k+1)/32);
    dim3 block(32,32);
    s2h<<<grid, block>>>(n/2, k, A+n/2, lda, Ah, n/2);
    s2h<<<grid, block>>>(n/2, k, A, lda, Bh, n/2);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n/2, n/2, k,
        &alpha, Ah, HIP_R_16F, n/2, Bh, HIP_R_16F, n/2,
        &beta, C+n/2, HIP_R_32F, ldc, HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );

    //float t = stopTimer();

    //printf("GEMM size m,n,k = %d %d %d,takes %lf, %lf TFLOPS\n", n/2, n/2, k, t, 2.0*n/2*n/2*k/1e9/t);

    //ge+=t;

    syrk(handle, n/2, k, alpha, A+n/2, lda, beta, C+n/2+ldc/2*n, ldc, hwork);

    //Transpose and cpy

}

void later_rsyrk(int n, int k,  float alpha, float *A, int lda, float beta, float *C, int ldc, __half *work)
{
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    startTimer();
    syrk(handle, n, k, alpha, A, lda, beta, C, ldc, work);
    printf("rsyrk takes %f ms\n", stopTimer());

    //printf("Panel takes %lf ms\n Gemm takes %lf ms\n", pan, ge);


    hipblasDestroy(handle);
}