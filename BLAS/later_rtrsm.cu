#include "hip/hip_runtime.h"
#include "LATER.h"

#include <hip/hip_fp16.h>

#define BLOCKSIZE 2048

float sone = 1.0;
float snegone = -1.0;
float szero = 0.0;

float panelTime = 0.0;
float gemmTime = 0.0;



void trsm(hipblasHandle_t handle, int m, int n, float* A, int lda, float* B, int ldb, __half* hwork)
{
    //printf("m,n=%d,%d\n", m, n);
    if(m <= BLOCKSIZE)
    {
        startTimer();
        hipblasStrsm(handle,
            HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
            m, n, &sone,
            A, lda,
            B, ldb
        );
        panelTime += stopTimer();
        //printf("%lf\n",panelTime);
        return;
    }
    trsm(handle, m/2, n, A, lda, B, ldb, hwork);
    
    __half *Ah = hwork;
    __half *Bh = hwork+m/2*m/2;

    dim3 grid((m/2+31)/32, (m/2+31)/32);
    dim3 block(32,32);
    s2h<<<grid, block>>>(m/2, m/2, A+m/2, lda, Ah, m/2);

    dim3 grid1((m/2+31)/32, (n+31)/32);
    dim3 block1(32,32);
    s2h<<<grid1, block1>>>(m/2, n, B, ldb, Bh, m/2);
    startTimer();

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m/2, n, m/2,
        &snegone, Ah, HIP_R_16F, m/2, Bh, HIP_R_16F, m/2,
        &sone, B+m/2, HIP_R_32F, ldb, HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );


    gemmTime +=stopTimer();
    //printf("%lf\n",gemmTime);
    //printMatrixDeviceBlock("ta.csv", m/2, m/2, A+m/2*m+m/2, lda);
    //printMatrixDeviceBlock("tb.csv", m/2, n, B+m/2, ldb);
    trsm(handle, m/2, n, A+m/2*lda+m/2, lda, B+m/2, ldb, hwork);
    //printf("1111111\n");
    //printMatrixDeviceBlock("tx.csv", m/2, n, B+m/2, ldb);
    
}

void later_rtrsm(int m, int n, float* A, int lda, float* B, int ldb, __half* hwork)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //printMatrixDeviceBlock("A.csv", m, m, A, lda);
    //printMatrixDeviceBlock("B.csv", m, n, B, ldb);
    trsm(handle, m, n, A, lda, B, ldb, hwork);
    printf("Panel takes %lfms\n", panelTime);
    printf("Gemm takes %lfms\n", gemmTime);
    //printf("22222\n");
    //printMatrixDeviceBlock("X.csv", m, n, B, ldb);

    hipblasDestroy(handle);

    return;
}

/*
void later_rtrsm(int m, int n, float* A, int lda, float* B, int ldb, __half* hwork)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int i = 0; i<m; i+=BLOCKSIZE)
    {
        int nb = min(m-i, BLOCKSIZE);

        startTimer();
        //leaf op
        hipblasStrsm(handle,
            HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
            nb, n, &sone,
            A+i, lda,
            B+i, ldb
        );

        panelTime+=stopTimer();

        startTimer();

        //if not the last block, then update
        if(m-i>BLOCKSIZE)
        {
            __half *Ah = hwork;
            __half *Bh = hwork + (m - i - nb)*nb;

            dim3 grid((m - i - nb + 31)/32, (nb+31)/32);
            dim3 block(32,32);
            s2h<<<grid, block>>>(m - i - nb, nb, A+i+nb, lda, Ah, m - i - nb);

            dim3 grid1((nb + 31)/32, (n+31)/32);
            dim3 block1(32,32);
            s2h<<<grid1, block1>>>(nb, n, B+i, ldb, Bh, nb);

            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m - i - nb, n, nb,
                &snegone, Ah, HIP_R_16F, m - i - nb, Bh, HIP_R_16F, nb,
                &sone, B+i+nb, HIP_R_32F, ldb, HIP_R_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP
            );
        }
        gemmTime+=stopTimer();
    }

    printf("Panel takes %lfms\n", panelTime);
    printf("Gemm takes %lfms\n", gemmTime);
}*/
