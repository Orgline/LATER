#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hiprand.h>
#include <stdlib.h>
#include <assert.h>
#include "LATER.h"

#ifdef MAGMA
#include "magma_v2.h"
#endif

int main(int argc, char *argv[]) 
{
#ifdef MAGMA
    printf("MAGMA LIB FOUND!\n");

#endif
    if (argc < 2) {
        printf(" usage: %s <n> \n", argv[0]); 
        return 0; 
    }
    int n = atoi(argv[1]); 


    hipsolverHandle_t cusolver_handle = NULL; 
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS; 

    cusolver_status = hipsolverDnCreate(&cusolver_handle); 
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status); 

    float *A, *D, *E, *tau; 
    //int n = 8192; 
    int lda = n; 
    hipMalloc( &A, sizeof(*A) * n * lda ); 
    hipMalloc( &D, sizeof(*A) * n  ); 
    hipMalloc( &E, sizeof(*A) * n  ); 
    hipMalloc( &tau, sizeof(*A) * n  ); 

    hiprandGenerator_t gen; 
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT); 

    {
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); 
        hiprandGenerateUniform(gen, A, n*lda); 

        int lwork;
        hipsolverDnSsytrd_bufferSize( cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, A, lda, D, E, tau,  &lwork);
        printf("n=%d, Ssytrd buffsize lwork=%d\n", n, lwork); 

        float *work;
        int *devInfo; 
        hipMalloc( &work, sizeof(*work)*lwork );
        hipMalloc( &devInfo, sizeof(*devInfo));
        startTimer();
        hipsolverDnSsytrd( cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, A, lda, D, E, tau, work, lwork, devInfo);
        float ms = stopTimer(); // in milliseconds
        int info; 
        hipMemcpy( &info, devInfo, sizeof(info), hipMemcpyDeviceToHost );
        float GFLOPS = 4.0/3.0 * n * n * n / (ms*1.0e6);
        printf(" Ssytrd info=%d took time %.0f milliseconds GFLOPS: %.0f\n", info, ms, GFLOPS );
    }
    {
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); 
        hiprandGenerateUniform(gen, A, n*lda); 

        int lwork;
        auto jobz = HIPSOLVER_EIG_MODE_NOVECTOR; 
        //auto jobz = HIPSOLVER_EIG_MODE_VECTOR; 
        hipsolverDnSsyevd_bufferSize( cusolver_handle, jobz, HIPBLAS_FILL_MODE_LOWER, n, A, lda, D,  &lwork);
        printf("n=%d, Ssyevd buffsize lwork=%d\n", n, lwork); 

        float *work;
        int *devInfo; 
        hipMalloc( &work, sizeof(*work)*lwork );
        hipMalloc( &devInfo, sizeof(*devInfo));
        startTimer();
        hipsolverDnSsyevd( cusolver_handle, jobz, HIPBLAS_FILL_MODE_LOWER, n, A, lda, D, work, lwork, devInfo);
        float ms = stopTimer(); // in milliseconds
        int info; 
        hipMemcpy( &info, devInfo, sizeof(info), hipMemcpyDeviceToHost );
        float GFLOPS = 4.0/3.0 * n * n * n / (ms*1.0e6);
        printf(" Ssyevd (novector) info=%d took time %.0f milliseconds GFLOPS: %.0f\n", info, ms, GFLOPS );
    }
    {
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); 
        hiprandGenerateUniform(gen, A, n*lda); 

        int lwork;
        auto jobz = HIPSOLVER_EIG_MODE_VECTOR; 
        //auto jobz = HIPSOLVER_EIG_MODE_VECTOR; 
        hipsolverDnSsyevd_bufferSize( cusolver_handle, jobz, HIPBLAS_FILL_MODE_LOWER, n, A, lda, D,  &lwork);
        printf("n=%d, Ssyevd buffsize lwork=%d\n", n, lwork); 

        float *work;
        int *devInfo; 
        hipMalloc( &work, sizeof(*work)*lwork );
        hipMalloc( &devInfo, sizeof(*devInfo));
        startTimer();
        hipsolverDnSsyevd( cusolver_handle, jobz, HIPBLAS_FILL_MODE_LOWER, n, A, lda, D, work, lwork, devInfo);
        float ms = stopTimer(); // in milliseconds
        int info; 
        hipMemcpy( &info, devInfo, sizeof(info), hipMemcpyDeviceToHost );
        float GFLOPS = 4.0/3.0 * n * n * n / (ms*1.0e6);
        printf(" Ssyevd (with vector) info=%d took time %.0f milliseconds GFLOPS: %.0f\n", info, ms, GFLOPS );
    }
#ifdef MAGMA
    {
        magma_print_environment();
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); 
        hiprandGenerateUniform(gen, A, n*lda); 
        magma_init();
        printf("hey magma worked! let's do some number crunching\n"); 
        int nb = 128; 
        int lwork = 2*n*nb; 
        int info; 

        float *hA = (float*) malloc( sizeof(*hA) * n * lda ); 
        hipMemcpy( hA, A, sizeof(*hA)*n*lda, hipMemcpyDeviceToHost ); 
        magma_ssytrd_sy2sb(MagmaLower, n, nb, A, lda, tau, work, lwork, &info); 
        magma_finalize();
    }
#endif
}
