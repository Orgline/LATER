#include "hip/hip_runtime.h"
#include "LATER.h"

hipEvent_t begin, end;
void startTimer()
{
    hipEventCreate(&begin);
    hipEventRecord(begin);
    hipEventCreate(&end);
}

float stopTimer()
{
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, begin, end);
    hipEventDestroy(begin);
    hipEventDestroy(end);
    return milliseconds;
}

__global__
void s2h(int m, int n, float *as, int ldas, __half *ah, int ldah)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		ah[i + j*ldah] = __float2half(as[i + j*ldas]);
	}
}

__global__
void h2s(int m, int n,__half *ah, int ldah, float *as, int ldas)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		as[i + j*ldah] = __half2float(ah[i + j*ldas]);
	}
}

void generateNormalMatrix(float *dA,int m,int n)
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    int seed = rand()%3000;
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateNormal(gen, dA, m*n,0,1);
}

void generateUniformMatrix(float *dA,int m,int n)
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    int seed = 3000;
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateUniform(gen,dA,m*n);
}

float snorm(int m,int n,float* dA)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float sn;
    int incx = 1;
    hipblasSnrm2(handle, m*n, dA, incx, &sn);
    hipblasDestroy(handle);
    return sn;
}

__global__
void setEye( int m, int n, float *a, int lda)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		if (i == j) 
			a[i+j*lda] = 1;
		else
			a[i+j*lda] = 0;
	}
}